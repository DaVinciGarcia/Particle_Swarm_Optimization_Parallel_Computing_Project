#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <cmath> 
#include <string>
#include <ctime> 
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>
#include <iomanip>
#include <hiprand/hiprand_kernel.h>
#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

const unsigned int parts_qty = 1000;
const unsigned int iterations = 1000;
const float min_range_value = -5.12f;
const float max_range_value = 5.12f;
const float w = 0.7f;
const float c1 = 1.5f;
const float c2 = 1.5f;

const int threadsPerBlock = 256;
const int blocksPerGrid = (parts_qty + threadsPerBlock - 1) / threadsPerBlock;

struct Particle {
    float* current_position_inx; 
    float* current_position_iny;

    float* best_position_inx; 
    float* best_position_iny;

    float* velocity_inx;
    float* velocity_iny;

    float* current_value;
    float* pBest;
};

__device__ float calcFunct(float pos_x, float pos_y) {
    return (20 + (pos_x * pos_x) + (pos_y * pos_y) - 
            10 * (cosf(2 * M_PI * pos_x) + cosf(2 * M_PI * pos_y)));
}

__global__ void initializeRandomPositions(float* position_x, 
                                          float* position_y, 
                                          float* velocity_x, 
                                          float* velocity_y,
                                          float* personal_best_x,
                                          float* personal_best_y, 
                                          int seed, float minVal, 
                                          float maxVal, int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    hiprandState state;
    hiprand_init(seed, tid, 0, &state);

    if (tid < N) { 
        position_x[tid] = minVal + (maxVal - minVal) * hiprand_uniform(&state);
        position_y[tid] = minVal + (maxVal - minVal) * hiprand_uniform(&state);
        velocity_x[tid] = minVal + (maxVal - minVal) * hiprand_uniform(&state);
        velocity_y[tid] = minVal + (maxVal - minVal) * hiprand_uniform(&state);
        personal_best_x[tid] = position_x[tid];
        personal_best_y[tid] = position_y[tid];
    }
}

__global__ void evalFunct(float* position_x, float* position_y, float* value) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < parts_qty) {
        value[tid] = calcFunct(position_x[tid], position_y[tid]);
    }
}

__global__ void copyTwoFloatValues(float* values_from, float* values_to, int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < N) {
        values_to[tid] = values_from[tid];
    }
}

__global__ void reduceMin(float* input, float* output, int* outputIndex, int n) {
    extern __shared__ float sdata[];
    int* sindex = (int*)&sdata[blockDim.x];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        sdata[tid] = input[i];
        sindex[tid] = i;
    } else {
        sdata[tid] = FLT_MAX;
        sindex[tid] = -1;
    }
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s && i + s < n) {
            if (sdata[tid] > sdata[tid + s]) {
                sdata[tid] = sdata[tid + s];
                sindex[tid] = sindex[tid + s];
            }
        }
        __syncthreads();
    }

    if (tid == 0) {
        output[blockIdx.x] = sdata[0];
        outputIndex[blockIdx.x] = sindex[0];
    }
}

__global__ void updateBestGlobal(float* personal_best, 
                                 float* global_best, 
                                 int* global_best_index, 
                                 int parts_qty) {
    *global_best = personal_best[0];
    *global_best_index = 0;
    for (int i = 1; i < parts_qty; i++) {
        if (personal_best[i] < *global_best) {
            *global_best = personal_best[i];
            *global_best_index = i;
        }
    }
}

__global__ void updateVelocity(float* d_current_position_inx,
                               float* d_current_position_iny,
                               float* d_best_position_inx,
                               float* d_best_position_iny,
                               float* d_velocity_inx,
                               float* d_velocity_iny,
                               int *d_team_best_index, 
                               float w, float c1, float c2, 
                               int parts_qty, hiprandState *state) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < parts_qty) {
        float r_ind = hiprand_uniform(state);
        float r_team = hiprand_uniform(state);
        d_velocity_inx[idx] = w * d_velocity_inx[idx] +
                              r_ind * c1 * (d_best_position_inx[idx] - d_current_position_inx[idx]) +
                              r_team * c2 * (d_best_position_inx[*d_team_best_index] - d_current_position_inx[idx]);

        d_velocity_iny[idx] = w * d_velocity_iny[idx] +
                              r_ind * c1 * (d_best_position_iny[idx] - d_current_position_iny[idx]) +
                              r_team * c2 * (d_best_position_iny[*d_team_best_index] - d_current_position_iny[idx]);
    }
}

__global__ void updatePosition(float* d_current_position_inx,
                               float* d_current_position_iny,
                               float* d_best_position_inx,
                               float* d_best_position_iny,
                               float* d_velocity_inx,
                               float* d_velocity_iny,
                               float* d_pBest,
                               int parts_qty) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < parts_qty) {
        d_current_position_inx[idx] += d_velocity_inx[idx];
        d_current_position_iny[idx] += d_velocity_iny[idx];

        float newValue = calcFunct(d_current_position_inx[idx], d_current_position_iny[idx]);
        if (newValue < d_pBest[idx]) {
            d_pBest[idx] = newValue;
            d_best_position_inx[idx] = d_current_position_inx[idx];
            d_best_position_iny[idx] = d_current_position_iny[idx];
        }
    }
}

int main() {

    Particle particle;

    particle.current_position_inx = new float[parts_qty];
    particle.current_position_iny = new float[parts_qty];
    particle.best_position_inx = new float[parts_qty];
    particle.best_position_iny = new float[parts_qty];
    particle.velocity_inx = new float[parts_qty];
    particle.velocity_iny = new float[parts_qty];
    particle.current_value = new float[parts_qty];
    particle.pBest = new float[parts_qty];
    float gBest;
    int gBestIndex;
    int h_best_global_indexes[blocksPerGrid];
    float h_best_global[blocksPerGrid];
    size_t sharedMemSize = threadsPerBlock * sizeof(float) + threadsPerBlock * sizeof(int);

    float* d_current_position_inx;
    float* d_current_position_iny;
    float* d_best_position_inx;
    float* d_best_position_iny;
    float* d_velocity_inx;
    float* d_velocity_iny;
    float* d_current_value;
    float* d_pBest;
    float* d_gBest;
    float* d_blocks_global_bests;
    int* d_blocks_global_bests_index;
    int* d_gBestIndex;
    hiprandState *state;

    hipMalloc((void**)&d_current_position_inx, parts_qty * sizeof(float));
    hipMalloc((void**)&d_current_position_iny, parts_qty * sizeof(float));
    hipMalloc((void**)&d_best_position_inx, parts_qty * sizeof(float));
    hipMalloc((void**)&d_best_position_iny, parts_qty * sizeof(float));
    hipMalloc((void**)&d_velocity_inx, parts_qty * sizeof(float));
    hipMalloc((void**)&d_velocity_iny, parts_qty * sizeof(float));
    hipMalloc((void**)&d_current_value, parts_qty * sizeof(float));
    hipMalloc((void**)&d_pBest, parts_qty * sizeof(float));
    hipMalloc((void**)&d_gBest, sizeof(float));
    hipMalloc((void**)&d_gBestIndex, sizeof(int));
    hipMalloc(&state, sizeof(hiprandState) * parts_qty);
    hipMalloc(&d_blocks_global_bests, blocksPerGrid * sizeof(float));
    hipMalloc(&d_blocks_global_bests_index, blocksPerGrid * sizeof(int));

    initializeRandomPositions<<<blocksPerGrid, threadsPerBlock>>>(d_current_position_inx, 
                                                                  d_current_position_iny, 
                                                                  d_velocity_inx, 
                                                                  d_velocity_iny,
                                                                  d_best_position_inx,
                                                                  d_best_position_iny, 
                                                                  time(NULL), 
                                                                  min_range_value, max_range_value,
                                                                  parts_qty);
    hipDeviceSynchronize();

    evalFunct<<<blocksPerGrid, threadsPerBlock>>>(d_current_position_inx, 
                                                  d_current_position_iny, 
                                                  d_current_value);
    hipDeviceSynchronize();

    copyTwoFloatValues<<<blocksPerGrid, threadsPerBlock>>>(d_current_value, d_pBest, parts_qty);
    hipDeviceSynchronize();

    updateBestGlobal<<<1,1>>>(d_pBest, d_gBest, d_gBestIndex, parts_qty);
    hipDeviceSynchronize();

    reduceMin<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_pBest, 
                                                                 d_blocks_global_bests, 
                                                                 d_blocks_global_bests_index, 
                                                                 parts_qty);
    hipDeviceSynchronize();

    hipMemcpy(h_best_global, d_blocks_global_bests, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_best_global_indexes, d_blocks_global_bests_index, blocksPerGrid * sizeof(int), hipMemcpyDeviceToHost);

    for (int k = 0; k < blocksPerGrid; k++) {
        std::cout << "Block " << k << " best value: " << h_best_global[k] << ", index: " << h_best_global_indexes[k] << std::endl;
    }

    for (int i = 0; i < iterations; i++) {
        updateVelocity<<<blocksPerGrid, threadsPerBlock>>>(d_current_position_inx,
                                                           d_current_position_iny,
                                                           d_best_position_inx,
                                                           d_best_position_iny,
                                                           d_velocity_inx,
                                                           d_velocity_iny,
                                                           d_gBestIndex, 
                                                           w, c1, c2, 
                                                           parts_qty, 
                                                           state);
        hipDeviceSynchronize();
        
        updatePosition<<<blocksPerGrid, threadsPerBlock>>>(d_current_position_inx,
                                                           d_current_position_iny,
                                                           d_best_position_inx,
                                                           d_best_position_iny,
                                                           d_velocity_inx,
                                                           d_velocity_iny,
                                                           d_pBest,
                                                           parts_qty);
        hipDeviceSynchronize();

        updateBestGlobal<<<1,1>>>(d_pBest, d_gBest, d_gBestIndex, parts_qty);
        hipDeviceSynchronize();
    }

    hipMemcpy(&gBest, d_gBest, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&gBestIndex, d_gBestIndex, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(particle.best_position_inx, d_best_position_inx, sizeof(float)*parts_qty, hipMemcpyDeviceToHost);
    hipMemcpy(particle.best_position_iny, d_best_position_iny, sizeof(float)*parts_qty, hipMemcpyDeviceToHost);

    std::cout << "Global best: " << std::fixed << std::setprecision(5) << gBest << std::endl;
    std::cout << "Global Best Position: (" << std::fixed << std::setprecision(5) << particle.best_position_inx[gBestIndex] << ", " 
              << particle.best_position_iny[gBestIndex] << ")" << std::endl;

    delete[] particle.current_position_inx;
    delete[] particle.current_position_iny;
    delete[] particle.velocity_inx;
    delete[] particle.velocity_iny;
    delete[] particle.current_value;
    delete[] particle.pBest;

    hipFree(d_current_position_inx);
    hipFree(d_current_position_iny);
    hipFree(d_best_position_inx);
    hipFree(d_best_position_iny);
    hipFree(d_velocity_inx);
    hipFree(d_velocity_iny);
    hipFree(d_current_value);
    hipFree(d_pBest);
    hipFree(d_gBest);
    hipFree(d_gBestIndex);
    hipFree(state);
    hipFree(d_blocks_global_bests);
    hipFree(d_blocks_global_bests_index);

    return 0;
}
