#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <cmath> 
#include <string>
#include <ctime> 
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>
#include <iomanip>
#include <hiprand/hiprand_kernel.h>
#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

const unsigned int parts_qty = 7500;
const unsigned int iterations = 1000;
const float min_range_value = -5.12f;
const float max_range_value = 5.12f;
const float w = 0.7f;
const float c1 = 1.5f;
const float c2 = 1.5f;

const int threadsPerBlock = 256;
const int blocksPerGrid = (parts_qty + threadsPerBlock - 1) / threadsPerBlock;

struct Particle {
    float* current_position_inx; 
    float* current_position_iny;

    float* best_position_inx; 
    float* best_position_iny;

    float* velocity_inx;
    float* velocity_iny;

    float* current_value;
    float* pBest;
};

__device__ float calcFunct(float pos_x, float pos_y) {
    return (20 + (pos_x * pos_x) + (pos_y * pos_y) - 
            10 * (cosf(2 * M_PI * pos_x) + cosf(2 * M_PI * pos_y)));
}

template <unsigned int blockSize>
__device__ void warpReduce(volatile float* sdata, volatile int* sindex, int tid) {
    if (blockSize >= 64) {
        if (sdata[tid] > sdata[tid + 32]) {
            sdata[tid] = sdata[tid + 32];
            sindex[tid] = sindex[tid + 32];
        }
    }
    if (blockSize >= 32) {
        if (sdata[tid] > sdata[tid + 16]) {
            sdata[tid] = sdata[tid + 16];
            sindex[tid] = sindex[tid + 16];
        }
    }
    if (blockSize >= 16) {
        if (sdata[tid] > sdata[tid + 8]) {
            sdata[tid] = sdata[tid + 8];
            sindex[tid] = sindex[tid + 8];
        }
    }
    if (blockSize >= 8) {
        if (sdata[tid] > sdata[tid + 4]) {
            sdata[tid] = sdata[tid + 4];
            sindex[tid] = sindex[tid + 4];
        }
    }
    if (blockSize >= 4) {
        if (sdata[tid] > sdata[tid + 2]) {
            sdata[tid] = sdata[tid + 2];
            sindex[tid] = sindex[tid + 2];
        }
    }
    if (blockSize >= 2) {
        if (sdata[tid] > sdata[tid + 1]) {
            sdata[tid] = sdata[tid + 1];
            sindex[tid] = sindex[tid + 1];
        }
    }
}

__global__ void initializeRandomPositions(float* position_x, 
                                          float* position_y, 
                                          float* velocity_x, 
                                          float* velocity_y,
                                          float* personal_best_x,
                                          float* personal_best_y, 
                                          int seed, float minVal, 
                                          float maxVal, int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    hiprandState state;
    hiprand_init(seed, tid, 0, &state);

    if (tid < N) { 
        position_x[tid] = minVal + (maxVal - minVal) * hiprand_uniform(&state);
        position_y[tid] = minVal + (maxVal - minVal) * hiprand_uniform(&state);
        velocity_x[tid] = minVal + (maxVal - minVal) * hiprand_uniform(&state);
        velocity_y[tid] = minVal + (maxVal - minVal) * hiprand_uniform(&state);
        personal_best_x[tid] = position_x[tid];
        personal_best_y[tid] = position_y[tid];
    }
}

__global__ void evalFunct(float* position_x, float* position_y, float* value) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < parts_qty) {
        value[tid] = calcFunct(position_x[tid], position_y[tid]);
    }
}

__global__ void copyTwoFloatValues(float* values_from, float* values_to, int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < N) {
        values_to[tid] = values_from[tid];
    }
}

template <unsigned int blockSize>
__global__ void reduceMin(float* input, float* output, int* outputIndex, int n) {
    extern __shared__ float sdata[];
    int* sindex = (int*)&sdata[blockDim.x];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockSize * 2) + threadIdx.x;
    unsigned int gridSize = blockSize * 2 * gridDim.x;

    sdata[tid] = FLT_MAX;
    sindex[tid] = -1;

    while (i < n) {
        float val1 = input[i];
        float val2 = (i + blockSize < n) ? input[i + blockSize] : FLT_MAX;

        if (val1 < sdata[tid]) {
            sdata[tid] = val1;
            sindex[tid] = i;
        }
        if (val2 < sdata[tid]) {
            sdata[tid] = val2;
            sindex[tid] = i + blockSize;
        }

        i += gridSize;
    }
    __syncthreads();

    // La reducción final dentro de la memoria compartida
    for (unsigned int s = blockSize / 2; s > 32; s >>= 1) {
        if (tid < s) {
            if (sdata[tid] > sdata[tid + s]) {
                sdata[tid] = sdata[tid + s];
                sindex[tid] = sindex[tid + s];
            }
        }
        __syncthreads();
    }

    if (tid < 32) warpReduce<blockSize>(sdata, sindex, tid);

    if (tid == 0) {
        output[blockIdx.x] = sdata[0];
        outputIndex[blockIdx.x] = sindex[0];
    }
}


__global__ void updateBestGlobal(float* personal_best,
                                    int* bests_index,
                                    float* global_best, 
                                    int* global_best_index, 
                                    int blocks) {
    *global_best = personal_best[0];
    *global_best_index = bests_index[0];
    for (int i = 1; i < blocks; i++) {
        if (personal_best[i] < *global_best) {
            *global_best_index = bests_index[i];
            *global_best = personal_best[i];
        }
    }
}

__global__ void updateVelocity(float* d_current_position_inx,
                               float* d_current_position_iny,
                               float* d_best_position_inx,
                               float* d_best_position_iny,
                               float* d_velocity_inx,
                               float* d_velocity_iny,
                               int *d_team_best_index, 
                               float w, float c1, float c2, 
                               int parts_qty, hiprandState *state) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < parts_qty) {
        float r_ind = hiprand_uniform(state);
        float r_team = hiprand_uniform(state);
        d_velocity_inx[idx] = w * d_velocity_inx[idx] +
                              r_ind * c1 * (d_best_position_inx[idx] - d_current_position_inx[idx]) +
                              r_team * c2 * (d_best_position_inx[*d_team_best_index] - d_current_position_inx[idx]);

        d_velocity_iny[idx] = w * d_velocity_iny[idx] +
                              r_ind * c1 * (d_best_position_iny[idx] - d_current_position_iny[idx]) +
                              r_team * c2 * (d_best_position_iny[*d_team_best_index] - d_current_position_iny[idx]);
    }
}

__global__ void updatePosition(float* d_current_position_inx,
                               float* d_current_position_iny,
                               float* d_best_position_inx,
                               float* d_best_position_iny,
                               float* d_velocity_inx,
                               float* d_velocity_iny,
                               float* d_pBest,
                               int parts_qty) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < parts_qty) {
        d_current_position_inx[idx] += d_velocity_inx[idx];
        d_current_position_iny[idx] += d_velocity_iny[idx];

        float newValue = calcFunct(d_current_position_inx[idx], d_current_position_iny[idx]);
        if (newValue < d_pBest[idx]) {
            d_pBest[idx] = newValue;
            d_best_position_inx[idx] = d_current_position_inx[idx];
            d_best_position_iny[idx] = d_current_position_iny[idx];
        }
    }
}

int main() {

    Particle particle;

    particle.current_position_inx = new float[parts_qty];
    particle.current_position_iny = new float[parts_qty];
    particle.best_position_inx = new float[parts_qty];
    particle.best_position_iny = new float[parts_qty];
    particle.velocity_inx = new float[parts_qty];
    particle.velocity_iny = new float[parts_qty];
    particle.current_value = new float[parts_qty];
    particle.pBest = new float[parts_qty];
    float gBest;
    int gBestIndex;
    int h_bests_global_indexes[blocksPerGrid];
    float h_bests_global[blocksPerGrid];
    size_t sharedMemSize = threadsPerBlock * sizeof(float) + threadsPerBlock * sizeof(int);

    float* d_current_position_inx;
    float* d_current_position_iny;
    float* d_best_position_inx;
    float* d_best_position_iny;
    float* d_velocity_inx;
    float* d_velocity_iny;
    float* d_current_value;
    float* d_pBest;
    float* d_gBest;
    float* d_blocks_global_bests;
    int* d_blocks_global_bests_index;
    int* d_gBestIndex;
    hiprandState *state;

    hipMalloc((void**)&d_current_position_inx, parts_qty * sizeof(float));
    hipMalloc((void**)&d_current_position_iny, parts_qty * sizeof(float));
    hipMalloc((void**)&d_best_position_inx, parts_qty * sizeof(float));
    hipMalloc((void**)&d_best_position_iny, parts_qty * sizeof(float));
    hipMalloc((void**)&d_velocity_inx, parts_qty * sizeof(float));
    hipMalloc((void**)&d_velocity_iny, parts_qty * sizeof(float));
    hipMalloc((void**)&d_current_value, parts_qty * sizeof(float));
    hipMalloc((void**)&d_pBest, parts_qty * sizeof(float));
    hipMalloc((void**)&d_gBest, sizeof(float));
    hipMalloc((void**)&d_gBestIndex, sizeof(int));
    hipMalloc(&state, sizeof(hiprandState) * parts_qty);
    hipMalloc(&d_blocks_global_bests, blocksPerGrid * sizeof(float));
    hipMalloc(&d_blocks_global_bests_index, blocksPerGrid * sizeof(int));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Registrar evento de inicio
    hipEventRecord(start);

    initializeRandomPositions<<<blocksPerGrid, threadsPerBlock>>>(d_current_position_inx, 
                                                                  d_current_position_iny, 
                                                                  d_velocity_inx, 
                                                                  d_velocity_iny,
                                                                  d_best_position_inx,
                                                                  d_best_position_iny, 
                                                                  time(NULL), 
                                                                  min_range_value, max_range_value,
                                                                  parts_qty);
    hipDeviceSynchronize();

    evalFunct<<<blocksPerGrid, threadsPerBlock>>>(d_current_position_inx, 
                                                  d_current_position_iny, 
                                                  d_current_value);
    hipDeviceSynchronize();

    // For initialize pBest = F(x,y)
    copyTwoFloatValues<<<blocksPerGrid, threadsPerBlock>>>(d_current_value, d_pBest, parts_qty);
    hipDeviceSynchronize();


    switch (threadsPerBlock)
    {
    case 512:
        reduceMin<512><<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_pBest, 
                                                                            d_blocks_global_bests, 
                                                                            d_blocks_global_bests_index, 
                                                                            parts_qty); break;
    case 256:
        reduceMin<256><<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_pBest, 
                                                                            d_blocks_global_bests, 
                                                                            d_blocks_global_bests_index, 
                                                                            parts_qty); break;
    case 128:
        reduceMin<128><<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_pBest, 
                                                                            d_blocks_global_bests, 
                                                                            d_blocks_global_bests_index, 
                                                                            parts_qty); break;
    case 64:
        reduceMin< 64><<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_pBest, 
                                                                            d_blocks_global_bests, 
                                                                            d_blocks_global_bests_index, 
                                                                            parts_qty); break;
    case 32:
        reduceMin< 32><<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_pBest, 
                                                                            d_blocks_global_bests, 
                                                                            d_blocks_global_bests_index, 
                                                                            parts_qty); break;
    case 16:
        reduceMin< 16><<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_pBest, 
                                                                            d_blocks_global_bests, 
                                                                            d_blocks_global_bests_index, 
                                                                            parts_qty); break;
    case 8:
        reduceMin< 8><<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_pBest, 
                                                                            d_blocks_global_bests, 
                                                                            d_blocks_global_bests_index, 
                                                                            parts_qty); break;
    case 4:
        reduceMin< 4><<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_pBest, 
                                                                            d_blocks_global_bests, 
                                                                            d_blocks_global_bests_index, 
                                                                            parts_qty); break;
    case 2:
        reduceMin< 2><<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_pBest, 
                                                                            d_blocks_global_bests, 
                                                                            d_blocks_global_bests_index, 
                                                                            parts_qty); break;
    case 1:
        reduceMin< 1><<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_pBest, 
                                                                            d_blocks_global_bests, 
                                                                            d_blocks_global_bests_index, 
                                                                            parts_qty); break;
    }
    hipDeviceSynchronize();
    
    hipMemcpy(h_bests_global, d_blocks_global_bests, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_bests_global_indexes, d_blocks_global_bests_index, blocksPerGrid * sizeof(int), hipMemcpyDeviceToHost);
    
    // for (int k = 0; k < blocksPerGrid; k++) {
    //     std::cout << "Block " << k << " best value: " << h_bests_global[k] << ", index: " << h_bests_global_indexes[k] << std::endl;
    // }

    //returns gBest and its index
    updateBestGlobal<<<1,1>>>(d_blocks_global_bests,
                                d_blocks_global_bests_index, 
                                d_gBest, 
                                d_gBestIndex, 
                                blocksPerGrid);

    hipDeviceSynchronize();

    hipMemcpy(&gBest, d_gBest, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&gBestIndex, d_gBestIndex, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(particle.best_position_inx, d_best_position_inx, sizeof(float)*parts_qty, hipMemcpyDeviceToHost);
    hipMemcpy(particle.best_position_iny, d_best_position_iny, sizeof(float)*parts_qty, hipMemcpyDeviceToHost);
    
    // std::cout << "After UpdateBestGlobal<<1,1>>: " <<  std::endl;
    // std::cout << "Global best: " << std::fixed << std::setprecision(5) << gBest << std::endl;
    // std::cout << "Global best index: " << std::fixed << std::setprecision(5) << gBestIndex << std::endl;
    // std::cout << "Global Best Position: (" << std::fixed << std::setprecision(5) << particle.best_position_inx[gBestIndex] << ", " 
    //           << particle.best_position_iny[gBestIndex] << ")" << std::endl;

    for (int i = 0; i < iterations; i++) {
        updateVelocity<<<blocksPerGrid, threadsPerBlock>>>(d_current_position_inx,
                                                           d_current_position_iny,
                                                           d_best_position_inx,
                                                           d_best_position_iny,
                                                           d_velocity_inx,
                                                           d_velocity_iny,
                                                           d_gBestIndex, 
                                                           w, c1, c2, 
                                                           parts_qty, 
                                                           state);
        hipDeviceSynchronize();
        
        updatePosition<<<blocksPerGrid, threadsPerBlock>>>(d_current_position_inx,
                                                           d_current_position_iny,
                                                           d_best_position_inx,
                                                           d_best_position_iny,
                                                           d_velocity_inx,
                                                           d_velocity_iny,
                                                           d_pBest,
                                                           parts_qty);
        hipDeviceSynchronize();

        switch (threadsPerBlock)
        {
        case 512:
            reduceMin<512><<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_pBest, 
                                                                                d_blocks_global_bests, 
                                                                                d_blocks_global_bests_index, 
                                                                                parts_qty); break;
        case 256:
            reduceMin<256><<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_pBest, 
                                                                                d_blocks_global_bests, 
                                                                                d_blocks_global_bests_index, 
                                                                                parts_qty); break;
        case 128:
            reduceMin<128><<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_pBest, 
                                                                                d_blocks_global_bests, 
                                                                                d_blocks_global_bests_index, 
                                                                                parts_qty); break;
        case 64:
            reduceMin< 64><<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_pBest, 
                                                                                d_blocks_global_bests, 
                                                                                d_blocks_global_bests_index, 
                                                                                parts_qty); break;
        case 32:
            reduceMin< 32><<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_pBest, 
                                                                                d_blocks_global_bests, 
                                                                                d_blocks_global_bests_index, 
                                                                                parts_qty); break;
        case 16:
            reduceMin< 16><<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_pBest, 
                                                                                d_blocks_global_bests, 
                                                                                d_blocks_global_bests_index, 
                                                                                parts_qty); break;
        case 8:
            reduceMin< 8><<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_pBest, 
                                                                                d_blocks_global_bests, 
                                                                                d_blocks_global_bests_index, 
                                                                                parts_qty); break;
        case 4:
            reduceMin< 4><<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_pBest, 
                                                                                d_blocks_global_bests, 
                                                                                d_blocks_global_bests_index, 
                                                                                parts_qty); break;
        case 2:
            reduceMin< 2><<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_pBest, 
                                                                                d_blocks_global_bests, 
                                                                                d_blocks_global_bests_index, 
                                                                                parts_qty); break;
        case 1:
            reduceMin< 1><<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_pBest, 
                                                                                d_blocks_global_bests, 
                                                                                d_blocks_global_bests_index, 
                                                                                parts_qty); break;
        }
        hipDeviceSynchronize();

        updateBestGlobal<<<1,1>>>(d_blocks_global_bests,
                                    d_blocks_global_bests_index, 
                                    d_gBest, 
                                    d_gBestIndex, 
                                    blocksPerGrid);
        hipDeviceSynchronize();

    }

    // Registrar evento de parada
    hipEventRecord(stop);

    // Esperar a que el evento de parada complete
    hipEventSynchronize(stop);

    // Calcular el tiempo de ejecución
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(&gBest, d_gBest, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&gBestIndex, d_gBestIndex, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(particle.best_position_inx, d_best_position_inx, sizeof(float)*parts_qty, hipMemcpyDeviceToHost);
    hipMemcpy(particle.best_position_iny, d_best_position_iny, sizeof(float)*parts_qty, hipMemcpyDeviceToHost);

    std::cout << "Global best: "<< gBest << " "; 
    std::cout << std::endl;
    std::cout << "Global Best Position: ("<< particle.best_position_inx[gBestIndex] << ", " 
                << particle.best_position_iny[gBestIndex] << ")"; 
    std::cout << std::endl;
    std::cout << "Global best index: "<< gBestIndex << " "; 
    std::cout << std::endl;
    std::cout << "Tiempo de ejecucion del kernel: " << milliseconds << " ms" << std::endl;


    delete[] particle.current_position_inx;
    delete[] particle.current_position_iny;
    delete[] particle.velocity_inx;
    delete[] particle.velocity_iny;
    delete[] particle.current_value;
    delete[] particle.pBest;

    hipFree(d_current_position_inx);
    hipFree(d_current_position_iny);
    hipFree(d_best_position_inx);
    hipFree(d_best_position_iny);
    hipFree(d_velocity_inx);
    hipFree(d_velocity_iny);
    hipFree(d_current_value);
    hipFree(d_pBest);
    hipFree(d_gBest);
    hipFree(d_gBestIndex);
    hipFree(state);
    hipFree(d_blocks_global_bests);
    hipFree(d_blocks_global_bests_index);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
